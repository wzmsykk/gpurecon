#include "hip/hip_runtime.h"
#include"headerfiles.h"
__global__ void Frotate(float * dst_image, float * src_image)
{
	for (int i=0;i<Nx;i++){
		for (int j = threadIdx.x+blockIdx.x*blockDim.x;j<Nz*Ny;j += blockDim.x * gridDim.x){
			dst_image[j+Ny*Nz*i] = src_image[(j%Ny)*Nz*Nx + (j/Ny)*Nx + i];
		}	
	}
}
//backimg==planeXZ
//backtempimg==planeYZ
//SRC[Y][Z][X]->DST[X][Z][Y]
//j%Ny=y j/Ny=z
//j=z*N_y+y
__global__ void Brotate(float* dst_image, float* src_image)
{
	for (int i = 0; i < Ny; i++) {
		for (int j = threadIdx.x + blockIdx.x * blockDim.x; j < Nz * Nx; j += blockDim.x * gridDim.x) {
			dst_image[j + Nz * Nx * i] = src_image[(j % Nx) * Nz * Ny + (j / Nx) * Ny + i];
		}
	}
}
//backimg==planeXZ
//backtempimg==planeYZ
//SRC[X][Z][Y]->DST[Y][Z][X]
//j%Nx=x j/Nx=z
//j=z*Nx+x
__global__ void Rrotate(float* dst_image, float* src_image) {
	for (int i = 0; i < Nz; i++) {
		for (int j = threadIdx.x + blockIdx.x * blockDim.x; j < Ny * Nx; j += blockDim.x * gridDim.x) {
			dst_image[j + Ny * Nx * i] = src_image[j % Nx + i * Nx + (j/Nx)*Nz*Nx];
		}
	}
}
//planeXZ->planeXY
//imageYZX==imageZYX
//SRC[Y][Z][X]->DST[Z][Y][X]
//i=z j%Nx=x j/Nx=y
//j=y*Nx+x
//