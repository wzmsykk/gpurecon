#include "hip/hip_runtime.h"
#include"headerfiles.h"
__global__ void genacmatrix(float* attenuation_matrix, CTdims* ctdim, short* ct_matrix) {
	//do nothing
	////miu=1e-8
	//AM[X][Y][Z]
	//CT[X][Y][Z]
	//����SIMPLEITK�ȹ��߽�CT����+ƽ����PETͼ��ͬXYZ��
	ctdim->x0 = -Nx * pixel_size/2.0f;
	ctdim->y0 = -Ny * pixel_size/2.0f;
	ctdim->z0 = -Nz * pixel_size / 2.0f;
	ctdim->xspacing = pixel_size;
	ctdim->yspacing = pixel_size;
	ctdim->zspacing = pixel_size;
	ctdim->xdim =Nx;
	ctdim->ydim =Ny;
	ctdim->zdim =Nz;
	for (int mx=0;mx< ctdim->xdim;mx++){
		for (int my = 0; my < ctdim->ydim; my++) {
			for (int mz = 0; mz < ctdim->zdim; mz++) {

				//short value = ct_matrix[mx*Ny*Nz+my*Nz+mz];
				double at = 0;
				//CT HU TO ATTENUATION LIST
				//H20 9.598E-02 cm-1=9.598E-3 mm-1
				//if (value<A && ){
				//atv=CONST01
				//}else if ..
				//TO DO 
				//����CT ��HUֵ����˥��
				//����ˮģ��2cm
				//�뾶5cm
				float rx = (mx - (ctdim->xdim - 1) / 2.0f) * ctdim->xspacing;
				float ry = (my - (ctdim->ydim - 1) / 2.0f) * ctdim->yspacing;
				float rz = (mz - (ctdim->zdim - 1) / 2.0f) * ctdim->zspacing;
				if ((rx * rx + ry * ry) < 2500.0f && rz <10.0f && rz>-10.0f) {
					attenuation_matrix[mz* ctdim->ydim * ctdim->xdim +my* ctdim->xdim +mx] = 9.598E-03*pixel_size ;
				}
				else {
					attenuation_matrix[mz * ctdim->ydim * ctdim->xdim + my * ctdim->xdim + mx] = 0;
				}//YZX

			}
		}

		
	}__syncthreads();

}