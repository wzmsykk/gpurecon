#include "hip/hip_runtime.h"
#include"headerfiles.h"

int genctdim(CTdims* host_ctdim, char* ctheader) {
	FILE* fp;
	char* linebuffer;
	char* p, * vp;
	int totalnumoflines;
	int cmpres = 0;
	if (strlen(ctheader) != 0) {
		fp = fopen(ctheader, "r");
		if (fp == nullptr) {
			printf("(ERROR) CT headerfile not found.\n");
			return -4;
		}
		totalnumoflines = GetLines(ctheader);
		printf("totallines=%d\n", totalnumoflines);
		linebuffer = (char*)malloc(sizeof(char) * 512);
		for (int i = 0; i < totalnumoflines; i++) {
			fgets(linebuffer, 512, fp);
			p = strtok(linebuffer, " ");
			if (!strcmp(p, "DimSize")) {
				p = strtok(NULL, "=");
				vp = strtok(p, " ");
				printf("vp=%s", vp);
				host_ctdim->xdim = atoi(vp);
				vp = strtok(NULL, " ");
				host_ctdim->ydim = atoi(vp);
				vp = strtok(NULL, " ");
				host_ctdim->zdim = atoi(vp);
			}
			else if (!strcmp(p, "Offset")) {
				p = strtok(NULL, "=");
				vp = strtok(p, " ");
				host_ctdim->x0 = atof(vp);
				vp = strtok(NULL, " ");
				host_ctdim->y0 = atof(vp);
				vp = strtok(NULL, " ");
				host_ctdim->z0 = atof(vp);
			}
			else if (!strcmp(p, "ElementSpacing")) {
				p = strtok(NULL, "=");
				vp = strtok(p, " ");
				host_ctdim->xspacing = atof(vp);
				vp = strtok(NULL, " ");
				host_ctdim->yspacing = atof(vp);
				vp = strtok(NULL, " ");
				host_ctdim->zspacing = atof(vp);
			}
		}
		fclose(fp);
		host_ctdim->x0 -= host_ctdim->xdim * host_ctdim->xspacing * 0.5;//������ԭ�����
		host_ctdim->y0 -= host_ctdim->ydim * host_ctdim->yspacing * 0.5;//������ԭ�����
		host_ctdim->z0 -= host_ctdim->zdim * host_ctdim->zspacing * 0.5;//������ԭ�����
	}
	else {
		host_ctdim->x0 = -Nx * pixel_size / 2.0f;
		host_ctdim->y0 = -Ny * pixel_size / 2.0f;
		host_ctdim->z0 = -Nz * pixel_size / 2.0f;
		host_ctdim->xspacing = pixel_size;
		host_ctdim->yspacing = pixel_size;
		host_ctdim->zspacing = pixel_size;
		host_ctdim->xdim = Nx;
		host_ctdim->ydim = Ny;
		host_ctdim->zdim = Nz;
	}
	printf("CT header read OK.\n");
	printf("(DEBUG) xoffset=%f,yoffset=%f,zoffset=%f\n", host_ctdim->x0, host_ctdim->y0, host_ctdim->z0);
	printf("(DEBUG) xspace=%f,yspace=%f,zspace=%f\n", host_ctdim->xspacing, host_ctdim->yspacing, host_ctdim->zspacing);
	printf("(DEBUG) xdim=%d,ydim=%d,zdim=%d\n", host_ctdim->xdim, host_ctdim->ydim, host_ctdim->zdim);

	return 0;
}
int genacmatrix(float* dev_attenuation_matrix, CTdims* host_ctdim, char* ct_bin_path, bool readCT) {
	CTdims* dev_ctdim;
	hipMalloc((void**)&dev_ctdim, sizeof(CTdims));
	size_t ctvoxcount = host_ctdim->xdim * host_ctdim->ydim * host_ctdim->zdim;
	short* dev_ct_matrix;
	hipMalloc((void**)&dev_ct_matrix, ctvoxcount * sizeof(short));
	short* host_ct_matrix = (short*)malloc(ctvoxcount * sizeof(short));


	if (readCT) {
		FILE* fp = fopen(ct_bin_path, "rb");
		if (fp == nullptr) {//�ļ��Ƿ����
			printf("(ERROR) CT binfile not found.\n");
			return -4;
		}
		size_t fsize_real, fsize_exp;
		fseek(fp, 0, SEEK_END);
		fsize_real = ftell(fp);
		
		fsize_exp = ctvoxcount * sizeof(short);//Ԥ����short TO DO ��������
		printf("(INFO) Expected bin file size:%zu, found bin file size:%zu", fsize_exp, fsize_real);
		if (fsize_exp != fsize_real) {
			printf("(ERROR) CT binfile size mismatch.\n");
			return -5;
		}
		fread(host_ct_matrix, sizeof(short), ctvoxcount, fp);
		fclose(fp);
		hipMemcpy(dev_ct_matrix, host_ct_matrix, ctvoxcount * sizeof(short), hipMemcpyHostToDevice);
		hipMemcpy(dev_ctdim, host_ctdim, sizeof(CTdims), hipMemcpyHostToDevice);
		genacvalue << <256, 512 >> > (dev_attenuation_matrix, dev_ctdim, dev_ct_matrix);
	}
	else {
		hipMemcpy(dev_ctdim, host_ctdim, sizeof(CTdims), hipMemcpyHostToDevice);
		genacvalue_fill <<<256, 512 >>> (dev_attenuation_matrix, dev_ctdim, dev_ct_matrix, 0.0);
		
	}
	
	

	
	

	



	
	hipDeviceSynchronize();
	free(host_ct_matrix);
	hipFree(dev_ct_matrix);
	return 0;
}
__global__ void genacvalue_fill(float* attenuation_matrix, CTdims* dev_ctdim, short* dev_ct_matrix, float value) {
	//gen a ac matrix filled by custom float
	int xdim = dev_ctdim->xdim;
	int ydim = dev_ctdim->ydim;
	int zdim = dev_ctdim->zdim;
	int totalvoxels = xdim * ydim * zdim;
	for (int line_index = threadIdx.x + blockIdx.x * blockDim.x; line_index < totalvoxels; line_index += blockDim.x * gridDim.x) {
		int mz = line_index / (xdim * ydim);
		int temp = line_index - mz * (xdim * ydim);
		int my = temp / xdim;
		int mx = temp - my * xdim;
		attenuation_matrix[mz * dev_ctdim->ydim * dev_ctdim->xdim + my * dev_ctdim->xdim + mx] = value;
	}

}



__global__ void genacvalue(float* attenuation_matrix, CTdims* dev_ctdim, short* dev_ct_matrix) {
	//do nothing
	////miu=1e-8
	//AM[X][Y][Z]
	//CT[X][Y][Z]
	//����SIMPLEITK�ȹ��߽�CT����+ƽ����PETͼ��ͬXYZ��
	int xdim = dev_ctdim->xdim;
	int ydim = dev_ctdim->ydim;
	int zdim = dev_ctdim->zdim;
	int totalvoxels = xdim * ydim * zdim;
	

	for (int line_index = threadIdx.x + blockIdx.x * blockDim.x; line_index < totalvoxels; line_index += blockDim.x * gridDim.x) {
		int mz = line_index / (xdim * ydim);
		int temp = line_index - mz * (xdim * ydim);
		int my = temp / xdim;
		int mx = temp - my * xdim;
		short ctvalue = dev_ct_matrix[line_index];
		//short value = dev_ct_matrix[mx*Ny*Nz+my*Nz+mz];
		//CT HU TO ATTENUATION LIST
		//H20 9.598E-02 cm-1=9.598E-3 mm-1
		//if (value<A && ){
		//atv=CONST01
		//}else if ..
		//TO DO 
		//����CT ��HUֵ����˥��
		//����ˮģ��2cm
		//�뾶5cm
		/*float rx = ((float)mx + 0.5) * dev_ctdim->xspacing + dev_ctdim->x0;
		float ry = ((float)my + 0.5) * dev_ctdim->yspacing + dev_ctdim->y0;
		float rz = ((float)mz + 0.5) * dev_ctdim->zspacing + dev_ctdim->z0;
		if ((rx * rx + ry * ry) < 2500.0f && rz <10.0f && rz>-10.0f) {
			attenuation_matrix[mz* dev_ctdim->ydim * dev_ctdim->xdim +my* dev_ctdim->xdim +mx] = 9.598E-03;
		}
		else {
			attenuation_matrix[mz * dev_ctdim->ydim * dev_ctdim->xdim + my * dev_ctdim->xdim + mx] = 0;
		}//YZX		*/
		if (dev_ct_matrix == nullptr) {
			attenuation_matrix[mz * dev_ctdim->ydim * dev_ctdim->xdim + my * dev_ctdim->xdim + mx] = 0;
		}
		else {
			if (ctvalue >= 0) {
				attenuation_matrix[mz * dev_ctdim->ydim * dev_ctdim->xdim + my * dev_ctdim->xdim + mx] = 0.1 * (0.096 + ctvalue * (0.172 - 0.096) / 1400.0);// unit per mm
			}
			else if (ctvalue < 0) {
				attenuation_matrix[mz * dev_ctdim->ydim * dev_ctdim->xdim + my * dev_ctdim->xdim + mx] = 0.1 * (ctvalue + 1000) * (0.096) / 1000.0;// unit per mm
			}
		}



	}__syncthreads();
}