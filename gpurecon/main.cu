#include "hip/hip_runtime.h"
#include "headerfiles.h"

#define GRIDSIZEX 128
#define BLOCKSIZEX 256



int main(int argc, char** argv)
{
//  to run:
//	nvcc -arch=sm_20 presort.cu 
//	./a.out will print usage
	hipError_t err = hipDeviceSetLimit(hipLimitMallocHeapSize, 1048576ULL * 256);
	//EXTREME BIG HEAP
	if(argc <=1)
	{
		printf("usage: [./a.out] [imageLORfilename] [normalizationLORfilename] [number of iteration] [batch size]\n");
		printf("if no normalization is available: \n");
		printf("usage: [./a.out] [imageLORfilename] [number of iteration] [batch size]\n");
		printf("then image is not normalized: may have ring gaps: \n");
		exit(1);
	}
	bool use_attu_corr = true;
	PrintConfig();

	int totalnumoflines,i;
	int shouldNormalize=0;
	int batchsize=128*128;
	double totalDeviceMemoryUsed=0;
	float * norm_image = (float *)malloc(sizeof(float)*Nx*Ny*Nz);
	float * dev_norm_image;
	hipMalloc ( ( void**)&dev_norm_image, Nx*Ny*Nz * sizeof(float) );
	totalDeviceMemoryUsed += (double)(4*Nx*Ny*Nz * sizeof(float));
	printf("(MEMORY): allocating normalization, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);
	int numoflinesForNorm=0;
	int iterationCount = 1;
	if(argc>4)
	{
		numoflinesForNorm=GetLines(argv[2]);
		printf("Calculating normalization image\n");
		shouldNormalize=1;
		iterationCount = atoi(argv[3]);
		batchsize=atoi(argv[4]);
		CalcNormImage(norm_image, numoflinesForNorm, argv[2]);

		FILE * save_norm_imagey;
		save_norm_imagey = fopen ("norm_image.bin" , "w");
		if (save_norm_imagey == NULL) 
		{
			printf("can not write to image file!\n");
			exit(1);
		}
		fwrite(norm_image, sizeof(float), Nx*Ny*Nz, save_norm_imagey);
		hipMemcpy(dev_norm_image, norm_image, Nx*Ny*Nz *sizeof(float ),hipMemcpyHostToDevice);
		free(norm_image);
	}
	else
	{
		if(argc>3)
		{
			iterationCount = atoi(argv[2]);
			batchsize=atoi(argv[3]);
		}
	}

	// get number of lines from lor files
	totalnumoflines=GetLines(argv[1]);
	if( totalnumoflines <= 0)
	{
		exit(1);
	}

	printf("Num of LORs is: %d\n",totalnumoflines);

	FILE * lor_data;
  	lor_data = fopen(argv[1], "r");
   	if (lor_data == NULL) {
		printf("lor data file not found\n");
		exit(1);
	}
	else 
	{
		printf("lor data file found as %s\n",argv[1]);
	}

	// read data from lor file:
	short *lor_data_array= (short *)malloc(sizeof(short) * totalnumoflines * 6);
	for (i=0;i<totalnumoflines;i++)
	{
		fscanf(lor_data,"%hd\t%hd\t%hd\t%hd\t%hd\t%hd\n",
			&lor_data_array[6*i],
			&lor_data_array[6*i+1],
			&lor_data_array[6*i+2],
			&lor_data_array[6*i+3],
			&lor_data_array[6*i+4],
			&lor_data_array[6*i+5]);
	}

	// copy data from local to device
	short *dev_lor_data_array;

	totalDeviceMemoryUsed += (double)(6*totalnumoflines * sizeof(short));
	printf("(MEMORY): allocating LOR data, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);

	hipMalloc ( ( void**)&dev_lor_data_array, 6*totalnumoflines * sizeof(short) );
	hipMemcpy(dev_lor_data_array, lor_data_array, 6*totalnumoflines *sizeof(short ),hipMemcpyHostToDevice);
	free(lor_data_array);
	
	float * dx_array; float * dy_array; float * dz_array;
	hipMalloc ( ( void**)&dx_array,totalnumoflines*sizeof(float));
	hipMalloc ( ( void**)&dy_array,totalnumoflines*sizeof(float));
	hipMalloc ( ( void**)&dz_array,totalnumoflines*sizeof(float));

	totalDeviceMemoryUsed += (double)(3*totalnumoflines * sizeof(float));
	printf("(MEMORY): allocating delta x, y, z data, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);
	printf("sorting delta x, delta y, delta z\n");
	convertolor<<<256,512>>>(dev_lor_data_array,dx_array,dy_array,dz_array,totalnumoflines);

	float *hx_array= (float *)malloc(sizeof(float)*totalnumoflines);
	float *hy_array= (float *)malloc(sizeof(float)*totalnumoflines);
	float *hz_array= (float *)malloc(sizeof(float)*totalnumoflines);	
	hipMemcpy(hx_array, dx_array, sizeof(float)*totalnumoflines,hipMemcpyDeviceToHost);
	hipMemcpy(hy_array, dy_array, sizeof(float)*totalnumoflines,hipMemcpyDeviceToHost);
	hipMemcpy(hz_array, dz_array, sizeof(float)*totalnumoflines,hipMemcpyDeviceToHost);
	hipFree(dx_array);hipFree(dy_array);hipFree(dz_array);

	totalDeviceMemoryUsed -= (double)(3*totalnumoflines * sizeof(float));
	printf("(MEMORY): de-allocating delta x, y, z data, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);

	int *indexxmax = (int *)malloc(sizeof(int)*totalnumoflines);
	int *indexymax = (int *)malloc(sizeof(int)*totalnumoflines);
	int *indexzmax = (int *)malloc(sizeof(int)*totalnumoflines);
	int *sizen = (int *)malloc(sizeof(int)*3);
	
	partlor(hx_array,hy_array,hz_array, totalnumoflines, indexxmax, indexymax, indexzmax, sizen);	
	free(hx_array);   free(hy_array);	free(hz_array);

	int *dev_indexxmax;	int *dev_indexymax;//	int *dev_indexzmax; 	
	hipMalloc ( ( void**)&dev_indexxmax, sizen[0] * sizeof(int) );
	hipMemcpy(dev_indexxmax, indexxmax, sizen[0] * sizeof(int),hipMemcpyHostToDevice);
	hipMalloc ( ( void**)&dev_indexymax, sizen[1] * sizeof(int) );
	hipMemcpy(dev_indexymax, indexymax, sizen[1] * sizeof(int),hipMemcpyHostToDevice);
	free(indexxmax);   free(indexymax);   free(indexzmax);   	

	totalDeviceMemoryUsed += (double)(sizen[0] * sizeof(int));
	totalDeviceMemoryUsed += (double)(sizen[1] * sizeof(int));
	printf("(MEMORY): allocating xz, yz plane max value, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);

	float *image = (float *)malloc(sizeof(float)*Nx*Ny*Nz);
	for (i=0;i<Nx*Ny*Nz;i++){*(image+i) = 1.0;}
	float * dev_image;
	hipMalloc ( ( void**)&dev_image, Nx*Ny*Nz * sizeof(float) );

	totalDeviceMemoryUsed += (double)(sizeof(float)*Nx*Ny*Nz);
	printf("(MEMORY): allocating output image, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);
	hipMemcpy(dev_image, image, Nx*Ny*Nz *sizeof(float ),hipMemcpyHostToDevice);
	free(image);
	
	float * host_back_image = (float *)malloc(sizeof(float)*Nx*Ny*Nz);
	for (i=0;i<Nx*Ny*Nz;i++){*(host_back_image+i) = 0;}
	float * dev_back_image;
	hipMalloc ( ( void**)&dev_back_image, Nx*Ny*Nz * sizeof(float) );

	float * dev_tempback_image;
	hipMalloc ( ( void**)&dev_tempback_image, Nx*Ny*Nz * sizeof(float) );

	totalDeviceMemoryUsed += (double)(2*sizeof(float)*Nx*Ny*Nz);
	printf("(MEMORY): allocating temp image for back projection, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);
	hipMemcpy(dev_tempback_image, host_back_image, Nx*Ny*Nz *sizeof(float ),hipMemcpyHostToDevice);
	hipMemcpy(dev_back_image, host_back_image, Nx*Ny*Nz *sizeof(float ),hipMemcpyHostToDevice);
	free(host_back_image);


	int nlines = 128*128; // can adjust this one to make recon faster (need more memory)
	nlines=batchsize;//in case of total events < default batchsize which caused blank image, change the batchsize to be less than total lines. 
	CUDAlor* lines;
	hipMalloc ( ( void**)&lines, sizeof(CUDAlor) * nlines );	// 11 elements for the lines structure

	totalDeviceMemoryUsed += (double)(sizeof(CUDAlor) * nlines );
	printf("(MEMORY): allocating memory to store temp lor data for forward projection, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);

	/*˥��������ʼ��*/
	float* device_attenuation_matrix; //�Դ�˥������
	CTdims* ctdim,*host_ctdim;			//CT����
	hipMalloc((void**)&ctdim, sizeof(CTdims));
	host_ctdim = (CTdims*)malloc(sizeof(CTdims));
	hipMemset(ctdim, 0, sizeof(CTdims));
	hipMalloc((void**)&device_attenuation_matrix, Nx* Ny* Nz * sizeof(float));
	hipMemset(device_attenuation_matrix,0, Nx * Ny * Nz * sizeof(float));

	totalDeviceMemoryUsed += (double)(2*sizeof(float) * Nx * Ny * Nz);
	printf("(MEMORY): allocating memory to store temp attenuation matrix, device memory used: %lf MB\n", totalDeviceMemoryUsed / 1048576.0);

	printf("(INFO): converting ct matrix values into attenuation values.\n");
	genacmatrix(device_attenuation_matrix,ctdim,nullptr);
	
	printf("(INFO): done.\n");
	if (DebugFile > 0) {
		SaveImageToFile(device_attenuation_matrix, "ATT_IMAGE.bin", Nx* Ny* Nz);//����˥�������ļ�
	}
	hipMemcpy(host_ctdim, ctdim, sizeof(CTdims), hipMemcpyDeviceToHost);

	//��������
	LineStatus* linestat; //LOR stat
	float* amin, * amax;  
	float* tempvec_x_4f, * tempvec_y_4f, * tempvec_z_4f;
	float* tempmat_alphas;
	float* mat_alphas;
	float* dis;
	int* alphavecsize;
	host_ctdim = (CTdims*)malloc(sizeof(CTdims));
	hipMemcpy(host_ctdim, ctdim, sizeof(CTdims), hipMemcpyDeviceToHost);

	int xdim = host_ctdim->xdim;
	int ydim = host_ctdim->ydim;
	int zdim = host_ctdim->zdim;
	int max_len = xdim + ydim + zdim + 3 + 2;
	free(host_ctdim);

	size_t onelinebuffersize = 0;
	int linesN = nlines;//ÿ����ͬʱ����linesN��
	VoxelID* voxelidvec;
	//�����Դ�
	hipMalloc((void**)&linestat, sizeof(LineStatus) * linesN);
	hipMemset((void*)linestat, 0, sizeof(LineStatus) * linesN);
	onelinebuffersize += sizeof(LineStatus);
	hipMalloc((void**)&tempvec_x_4f, sizeof(float) * linesN * 4);
	onelinebuffersize += sizeof(float);
	hipMalloc((void**)&tempvec_y_4f, sizeof(float) * linesN * 4);
	onelinebuffersize += sizeof(float);
	hipMalloc((void**)&tempvec_z_4f, sizeof(float) * linesN * 4);
	onelinebuffersize += sizeof(float);
	hipMalloc((void**)&amin, sizeof(float) * linesN);
	onelinebuffersize += sizeof(float);
	hipMalloc((void**)&amax, sizeof(float) * linesN);
	onelinebuffersize += sizeof(float);
	hipMalloc((void**)&tempmat_alphas, sizeof(float) * linesN * max_len);
	hipMemset((void*)tempmat_alphas, 0, sizeof(float) * linesN * max_len);
	onelinebuffersize += sizeof(float) * max_len;
	hipMalloc((void**)&voxelidvec, sizeof(VoxelID) * linesN * max_len);
	onelinebuffersize += sizeof(VoxelID) * max_len;
	hipMalloc((void**)&dis, sizeof(float) * linesN * max_len);
	onelinebuffersize += sizeof(float) * max_len;
	hipMalloc((void**)&alphavecsize, sizeof(int) * linesN);
	hipMemset((void*)alphavecsize, 0, sizeof(int) * linesN);
	onelinebuffersize += sizeof(int);
	hipMalloc((void**)&mat_alphas, sizeof(float) * linesN * max_len);
	hipMemset((void*)mat_alphas, 0, sizeof(float) * linesN * max_len);
	onelinebuffersize += sizeof(float) * max_len;
	hipDeviceSynchronize();

	totalDeviceMemoryUsed += (double)(onelinebuffersize* linesN);
	printf("(MEMORY): allocating memory to for attenuation calculation, device memory used: %lf MB\n", totalDeviceMemoryUsed / 1048576.0);

	/*˥��������ʼ������*/


	int totalnumoflinesxz = sizen[1];
	int totalnumoflinesyz = sizen[0];

	printf("\nlor memory are prepared now running OSEM (running batches of %d lors) \n\n", nlines);

	






	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	printf("total iteration: #%d\n",iterationCount);


	int maxxzbatch, maxyzbatch;
	int realnlines;


	//����˥������
	float* linesxz_attvalue_list, *linesyz_attvalue_list;
	hipMalloc((void**)&linesxz_attvalue_list, sizeof(float) * totalnumoflinesxz);
	hipMalloc((void**)&linesyz_attvalue_list, sizeof(float) * totalnumoflinesyz);
	maxxzbatch = ceil(totalnumoflinesxz / (float)nlines);
	maxyzbatch = ceil(totalnumoflinesyz / (float)nlines);
	//����XZ���ϵ�˥��ֵ
	printf("Doing attu corr for XZ lines.\n");
	for (i = 0; i < maxxzbatch; i++) {
		realnlines = nlines;
		if ((i + 1) * nlines > totalnumoflinesxz) {
			realnlines = totalnumoflinesxz - i * nlines;
			printf("(DEBUG) LAST BATCH XZ LOR SIZE=%d\n", realnlines);
		}//��ֹ��������batchsize batchsize<0������ֵ�bug
		int noffset = i * nlines;
		convertolorxz << <256, 512 >> > (dev_lor_data_array, dev_indexymax, lines, realnlines, noffset);//��lor_data��lor����index_y_max����lines
		
		//˥��������ʼ
		linesN = realnlines;//������ʵ�ʵ�LOR���� realnlines<=linesN

		calc_stat << <GRIDSIZEX, BLOCKSIZEX >> > (lines, linesN, linestat);
		alphaextrema << <GRIDSIZEX, BLOCKSIZEX >> > (lines, linesN, ctdim, linestat, amin, amax, tempvec_x_4f, tempvec_y_4f);
		alphavecs << <GRIDSIZEX, BLOCKSIZEX >> > (lines, linesN, ctdim, linestat, amin, amax, tempmat_alphas, mat_alphas, alphavecsize);
		dist_and_ID_in_voxel << <GRIDSIZEX, BLOCKSIZEX >> > (lines, linesN, ctdim, linestat, voxelidvec, dis, mat_alphas, alphavecsize);
		attu_inner_product << <GRIDSIZEX, BLOCKSIZEX >> > (lines, linesN, ctdim, device_attenuation_matrix, linestat, voxelidvec, dis, alphavecsize);


		extract_attenu_value_to_list_with_offset << <GRIDSIZEX, BLOCKSIZEX >> > (lines, linesN, linesxz_attvalue_list, noffset);
		//�������
		hipMemset((void*)linestat, 0, sizeof(LineStatus) * linesN);
		hipMemset((void*)tempmat_alphas, 0, sizeof(float) * linesN * max_len);
		hipMemset((void*)alphavecsize, 0, sizeof(int) * linesN);
		hipMemset((void*)mat_alphas, 0, sizeof(float) * linesN * max_len);
		hipDeviceSynchronize();

		//˥����������

	}
	printf("attu corr for XZ lines ends.\n");
	//����YZ���ϵ�˥��ֵ
	printf("Doing attu corr for YZ lines.\n");
	for (i = 0; i < maxyzbatch; i++) {
		realnlines = nlines;
		if ((i + 1) * nlines > totalnumoflinesyz) {
			realnlines = totalnumoflinesyz - i * nlines;
			printf("(DEBUG) LAST BATCH YZ LOR SIZE=%d\n", realnlines);
		}//��ֹ��������batchsize batchsize<0������ֵ�bug
		int noffset = i * nlines;
		convertolorxz << <256, 512 >> > (dev_lor_data_array, dev_indexxmax, lines, realnlines, noffset);//��lor_data��lor����index_x_max����lines

		//˥��������ʼ
		linesN = realnlines;//������ʵ�ʵ�LOR���� realnlines<=linesN

		calc_stat << <GRIDSIZEX, BLOCKSIZEX >> > (lines, linesN, linestat);
		alphaextrema << <GRIDSIZEX, BLOCKSIZEX >> > (lines, linesN, ctdim, linestat, amin, amax, tempvec_x_4f, tempvec_y_4f);
		alphavecs << <GRIDSIZEX, BLOCKSIZEX >> > (lines, linesN, ctdim, linestat, amin, amax, tempmat_alphas, mat_alphas, alphavecsize);
		dist_and_ID_in_voxel << <GRIDSIZEX, BLOCKSIZEX >> > (lines, linesN, ctdim, linestat, voxelidvec, dis, mat_alphas, alphavecsize);
		attu_inner_product << <GRIDSIZEX, BLOCKSIZEX >> > (lines, linesN, ctdim, device_attenuation_matrix, linestat, voxelidvec, dis, alphavecsize);


		extract_attenu_value_to_list_with_offset << <GRIDSIZEX, BLOCKSIZEX >> > (lines, linesN, linesyz_attvalue_list, noffset);
		//�������
		hipMemset((void*)linestat, 0, sizeof(LineStatus) * linesN);
		hipMemset((void*)tempmat_alphas, 0, sizeof(float) * linesN * max_len);
		hipMemset((void*)alphavecsize, 0, sizeof(int) * linesN);
		hipMemset((void*)mat_alphas, 0, sizeof(float) * linesN * max_len);
		hipDeviceSynchronize();

		//˥����������

	}
	printf("attu corr for YZ lines ends.\n");



	//������ά�ؽ�
	for (int iter=0;iter<iterationCount;iter++)
	{
		printf("now iteration: #%d\n", iter);
		if (DebugInfo > 0)
		{
			printf("***********************************************************************************\n");
			printf("Doing forward and backward projection for plane xz with lor hitting xz plane (lor-xz)\n");
			printf("***********************************************************************************\n");
		}

		//TO DO 
		maxxzbatch = ceil(totalnumoflinesxz / (float)nlines);
		//maxxzbatch = 1;
		for (i= 0; i< maxxzbatch; i++)
		{
			
			realnlines = nlines;
			//realnlines = 3;
			if ((i+1) * nlines > totalnumoflinesxz) {
				realnlines = totalnumoflinesxz - i * nlines;
				printf("(DEBUG) LAST BATCH XZ LOR SIZE=%d\n",realnlines);
			}//��ֹ��������batchsize batchsize<0������ֵ�bug
			int noffset = i*nlines;
			
			

			if (use_attu_corr) {
				
				convertolorxz_ac << <256, 512 >> > (dev_lor_data_array, dev_indexymax, lines, linesxz_attvalue_list, realnlines, noffset);//��lor_data��lor����index_y����lines
				Forwardprojxz << <256, 512 >> > (dev_image, lines, realnlines);
				Backprojxz_ac << <256, 512 >> > (dev_image, dev_back_image, lines, realnlines, 0);//changed 			
			}					//Backprojxz<<<128,128>>>(dev_image,dev_back_image,lines,realnlines,0);
			else {
				convertolorxz << <256, 512 >> > (dev_lor_data_array, dev_indexymax, lines, realnlines, noffset);//��lor_data��lor����index_y����lines
				Forwardprojxz << <256, 512 >> > (dev_image, lines, realnlines);
				Backprojxz << <256, 512 >> > (dev_image, dev_back_image, lines, realnlines, 0);//changed
			}
		} // if using OSEM, move the iteration to #OSEM
	
		if(DebugInfo>0)
		{
			printf("(IMAGE) rotated image 90 degrees to point to yz plane\n");
		}
		Frotate<<<256,512>>>(dev_back_image, dev_tempback_image);
		if (DebugFile > 0)
		{
			SaveImageToFile(dev_back_image, "dev_back_img.bin", Nx * Ny * Nz);
			SaveImageToFile(dev_tempback_image, "dev_back_img_roted.bin", Nx * Ny * Nz);
		}
		
		if(DebugInfo>0)
		{
			printf("***********************************************************************************\n");
			printf("Doing forward and backward projection for plane yz with lor hitting yz plane (lor-yz)\n");	
			printf("***********************************************************************************\n");
		}
		hipMemcpy(dev_back_image, dev_tempback_image, Nx*Ny*Nz *sizeof(float ),hipMemcpyDeviceToDevice);

		maxyzbatch = ceil(totalnumoflinesyz / (float)nlines);
		//maxyzbatch = 1;
		for (i = 0; i < maxyzbatch; i++)
		//for (i = 0; i < totalnumoflinesyz / nlines ; i++)
		{
			
			int realnlines = nlines;
			if ((i + 1) * nlines > totalnumoflinesyz) {
				realnlines = totalnumoflinesyz - i * nlines;
				printf("(DEBUG) LAST BATCH YZ LOR SIZE=%d\n", realnlines);
			}//��ֹ��������batchsize
			int noffset = i*nlines;
			

			

			if (use_attu_corr) {

				convertoloryz_ac << <256, 512 >> > (dev_lor_data_array, dev_indexxmax, lines, linesyz_attvalue_list, realnlines, noffset);
				Forwardprojyz << <256, 512 >> > (dev_image, lines, realnlines);
				Backprojyz_ac << <256, 512 >> > (dev_image, dev_back_image, lines, realnlines, 0);
			}
			else {
				convertoloryz << <256, 512 >> > (dev_lor_data_array, dev_indexxmax, lines, realnlines, noffset);
				Forwardprojyz << <256, 512 >> > (dev_image, lines, realnlines);
				Backprojyz << <256, 512 >> > (dev_image, dev_back_image, lines, realnlines, 0);
			}
			
			
		} // if using OSEM, move the iteration to #OSEM

		Brotate<<<256,512>>>(dev_back_image, dev_tempback_image);//ת��ȥ
		hipMemcpy(dev_back_image, dev_tempback_image, Nx*Ny*Nz *sizeof(float ),hipMemcpyDeviceToDevice);

		if(shouldNormalize>0)
		{
			Fnorm<<<17,720>>>(dev_image,dev_back_image,dev_norm_image);
		}
		else
		{
			hipMemcpy(dev_image, dev_back_image, Nx*Ny*Nz *sizeof(float ),hipMemcpyDeviceToDevice);
		}

		hipMemset(dev_back_image, 0, Nx*Ny*Nz *sizeof(float ));
		hipMemset(lines, 0, sizeof(CUDAlor) * nlines );
		// #OSEM (indicating OSEM)
	}

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,start,stop);


	SaveImageToFile(dev_image, "image.bin", Nx * Ny * Nz);
	Rrotate << <256, 512 >> > (dev_image, dev_tempback_image);//��ΪZYX��ʽ	
	SaveImageToFile(dev_tempback_image, "imageZYX.bin", Nx * Ny * Nz);
	printf("************************************************\n");
	printf("   all done!! elapsed time is %f s\n",elapsedTime/1000.0);	
	printf("************************************************\n");

	
	hipFree(dev_lor_data_array);
	hipFree(device_attenuation_matrix);
	hipFree(dev_image); hipFree(dev_back_image); hipFree(dev_tempback_image); hipFree(lines);
	hipFree(dev_indexxmax); hipFree(dev_indexymax);free(sizen);//hipFree(dev_indexzmax);



	//˥���������
	hipFree(linestat);
	hipFree(tempvec_x_4f);
	hipFree(tempvec_y_4f);
	hipFree(tempvec_z_4f);
	hipFree(amin);
	hipFree(amax);
	hipFree(tempmat_alphas);
	hipFree(voxelidvec);
	hipFree(dis);
	hipFree(alphavecsize);
	hipFree(mat_alphas);
	//����


	return 0;
}
