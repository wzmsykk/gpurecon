#include "hip/hip_runtime.h"
#include "headerfiles.h"

int main(int argc, char** argv)
{
//  to run:
//	nvcc -arch=sm_20 presort.cu 
//	./a.out will print usage

	if(argc <=1)
	{
		printf("usage: [./a.out] [imageLORfilename] [normalizationLORfilename] [number of iteration] [batch size]\n");
		printf("if no normalization is available: \n");
		printf("usage: [./a.out] [imageLORfilename] [number of iteration] [batch size]\n");
		printf("then image is not normalized: may have ring gaps: \n");
		exit(1);
	}

	PrintConfig();

	int totalnumoflines,i;
	int shouldNormalize=0;
	int batchsize=16384;
	double totalDeviceMemoryUsed=0;
	float * norm_image = (float *)malloc(sizeof(float)*Nx*Ny*Nz);
	float * dev_norm_image;
	hipMalloc ( ( void**)&dev_norm_image, Nx*Ny*Nz * sizeof(float) );
	totalDeviceMemoryUsed += (double)(4*Nx*Ny*Nz * sizeof(float));
	printf("(MEMORY): allocating normalization, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);
	int numoflinesForNorm=0;
	int iterationCount = 1;
	if(argc>4)
	{
		numoflinesForNorm=GetLines(argv[2]);
		printf("Calculating normalization image\n");
		shouldNormalize=1;
		iterationCount = atoi(argv[3]);
		batchsize=atoi(argv[4]);
		CalcNormImage(norm_image, numoflinesForNorm, argv[2]);

		FILE * save_norm_imagey;
		save_norm_imagey = fopen ("norm_image.bin" , "w");
		if (save_norm_imagey == NULL) 
		{
			printf("can not write to image file!\n");
			exit(1);
		}
		fwrite(norm_image, sizeof(float), Nx*Ny*Nz, save_norm_imagey);
		hipMemcpy(dev_norm_image, norm_image, Nx*Ny*Nz *sizeof(float ),hipMemcpyHostToDevice);
		free(norm_image);
	}
	else
	{
		if(argc>3)
		{
			iterationCount = atoi(argv[2]);
			batchsize=atoi(argv[3]);
		}
	}

	// get number of lines from lor files
	totalnumoflines=GetLines(argv[1]);
	if( totalnumoflines <= 0)
	{
		exit(1);
	}

	printf("Num of LORs is: %d\n",totalnumoflines);

	FILE * lor_data;
  	lor_data = fopen(argv[1], "r");
   	if (lor_data == NULL) {
		printf("lor data file not found\n");
		exit(1);
	}
	else 
	{
		printf("lor data file found as %s\n",argv[1]);
	}

	// read data from lor file:
	short *lor_data_array= (short *)malloc(sizeof(short) * totalnumoflines * 6);
	for (i=0;i<totalnumoflines;i++)
	{
		fscanf(lor_data,"%hd\t%hd\t%hd\t%hd\t%hd\t%hd\n",
			&lor_data_array[6*i],
			&lor_data_array[6*i+1],
			&lor_data_array[6*i+2],
			&lor_data_array[6*i+3],
			&lor_data_array[6*i+4],
			&lor_data_array[6*i+5]);
	}

	// copy data from local to device
	short *dev_lor_data_array;

	totalDeviceMemoryUsed += (double)(6*totalnumoflines * sizeof(short));
	printf("(MEMORY): allocating LOR data, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);

	hipMalloc ( ( void**)&dev_lor_data_array, 6*totalnumoflines * sizeof(short) );
	hipMemcpy(dev_lor_data_array, lor_data_array, 6*totalnumoflines *sizeof(short ),hipMemcpyHostToDevice);
	free(lor_data_array);
	
	float * dx_array; float * dy_array; float * dz_array;
	hipMalloc ( ( void**)&dx_array,totalnumoflines*sizeof(float));
	hipMalloc ( ( void**)&dy_array,totalnumoflines*sizeof(float));
	hipMalloc ( ( void**)&dz_array,totalnumoflines*sizeof(float));

	totalDeviceMemoryUsed += (double)(3*totalnumoflines * sizeof(float));
	printf("(MEMORY): allocating delta x, y, z data, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);
	printf("sorting delta x, delta y, delta z\n");
	convertolor<<<512,512>>>(dev_lor_data_array,dx_array,dy_array,dz_array,totalnumoflines);

	float *hx_array= (float *)malloc(sizeof(float)*totalnumoflines);
	float *hy_array= (float *)malloc(sizeof(float)*totalnumoflines);
	float *hz_array= (float *)malloc(sizeof(float)*totalnumoflines);	
	hipMemcpy(hx_array, dx_array, sizeof(float)*totalnumoflines,hipMemcpyDeviceToHost);
	hipMemcpy(hy_array, dy_array, sizeof(float)*totalnumoflines,hipMemcpyDeviceToHost);
	hipMemcpy(hz_array, dz_array, sizeof(float)*totalnumoflines,hipMemcpyDeviceToHost);
	hipFree(dx_array);hipFree(dy_array);hipFree(dz_array);

	totalDeviceMemoryUsed -= (double)(3*totalnumoflines * sizeof(float));
	printf("(MEMORY): de-allocating delta x, y, z data, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);

	int *indexxmax = (int *)malloc(sizeof(int)*totalnumoflines);
	int *indexymax = (int *)malloc(sizeof(int)*totalnumoflines);
	int *indexzmax = (int *)malloc(sizeof(int)*totalnumoflines);
	int *sizen = (int *)malloc(sizeof(int)*3);
	
	partlor(hx_array,hy_array,hz_array, totalnumoflines, indexxmax, indexymax, indexzmax, sizen);	
	free(hx_array);   free(hy_array);	free(hz_array);

	int *dev_indexxmax;	int *dev_indexymax;//	int *dev_indexzmax; 	
	hipMalloc ( ( void**)&dev_indexxmax, sizen[0] * sizeof(int) );
	hipMemcpy(dev_indexxmax, indexxmax, sizen[0] * sizeof(int),hipMemcpyHostToDevice);
	hipMalloc ( ( void**)&dev_indexymax, sizen[1] * sizeof(int) );
	hipMemcpy(dev_indexymax, indexymax, sizen[1] * sizeof(int),hipMemcpyHostToDevice);
	free(indexxmax);   free(indexymax);   free(indexzmax);   	

	totalDeviceMemoryUsed += (double)(sizen[0] * sizeof(int));
	totalDeviceMemoryUsed += (double)(sizen[1] * sizeof(int));
	printf("(MEMORY): allocating xz, yz plane max value, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);

	float *image = (float *)malloc(sizeof(float)*Nx*Ny*Nz);
	for (i=0;i<Nx*Ny*Nz;i++){*(image+i) = 1.0;}
	float * dev_image;
	hipMalloc ( ( void**)&dev_image, Nx*Ny*Nz * sizeof(float) );

	totalDeviceMemoryUsed += (double)(sizeof(float)*Nx*Ny*Nz);
	printf("(MEMORY): allocating output image, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);
	hipMemcpy(dev_image, image, Nx*Ny*Nz *sizeof(float ),hipMemcpyHostToDevice);
	free(image);
	
	float * host_back_image = (float *)malloc(sizeof(float)*Nx*Ny*Nz);
	for (i=0;i<Nx*Ny*Nz;i++){*(host_back_image+i) = 0;}
	float * dev_back_image;
	hipMalloc ( ( void**)&dev_back_image, Nx*Ny*Nz * sizeof(float) );

	float * dev_tempback_image;
	hipMalloc ( ( void**)&dev_tempback_image, Nx*Ny*Nz * sizeof(float) );

	totalDeviceMemoryUsed += (double)(2*sizeof(float)*Nx*Ny*Nz);
	printf("(MEMORY): allocating temp image for back projection, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);
	hipMemcpy(dev_tempback_image, host_back_image, Nx*Ny*Nz *sizeof(float ),hipMemcpyHostToDevice);
	hipMemcpy(dev_back_image, host_back_image, Nx*Ny*Nz *sizeof(float ),hipMemcpyHostToDevice);
	free(host_back_image);


	int nlines = 256*512; // can adjust this one to make recon faster (need more memory)
	nlines=batchsize;//in case of total events < default batchsize which caused blank image, change the batchsize to be less than total lines. 
	float * lines;
	hipMalloc ( ( void**)&lines, CUDAlor_size * nlines * sizeof(float) );	// 11 elements for the lines structure

	totalDeviceMemoryUsed += (double)(CUDAlor_size * nlines * sizeof(float));
	printf("(MEMORY): allocating memory to store temp lor data for forward projection, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);

	//new lines for attenuation correction
	float* attenuation_matrix;
	hipMalloc((void**)&attenuation_matrix, Nx* Ny* Nz * sizeof(float));
	hipMemset((void**)&attenuation_matrix, 0.0, Nx * Ny * Nz * sizeof(float));
	float* temp_attenuation_matrix;
	hipMalloc((void**)&temp_attenuation_matrix, Nx * Ny * Nz * sizeof(float));

	totalDeviceMemoryUsed += (double)(2*sizeof(float) * Nx * Ny * Nz);
	printf("(MEMORY): allocating memory to store temp attenuation matrix, device memory used: %lf MB\n", totalDeviceMemoryUsed / 1048576.0);


	genacmatrix<<<256,512>>>(attenuation_matrix);

	//new lines end

	int totalnumoflinesxz = sizen[1];
	int totalnumoflinesyz = sizen[0];

	printf("\nlor memory are prepared now running OSEM (running batches of %d lors) \n\n", nlines);
	if(DebugInfo>0)
	{
		printf("***********************************************************************************\n");
		printf("Doing forward and backward projection for plane xz with lor hitting xz plane (lor-xz)\n");	
		printf("***********************************************************************************\n");
	}

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	printf("total iteration: #%d\n",iterationCount);

	for (int iter=0;iter<iterationCount;iter++)
	{

		for (i=0; i<totalnumoflinesxz/nlines; i++)
		{
			int realnlines = nlines;
			int noffset = i*nlines;
			
			

			convertolorxz<<<256,512>>>(dev_lor_data_array,dev_indexymax,lines,realnlines,noffset);

			attenucorrxz << <256, 512 >> > (lines, realnlines, attenuation_matrix);//new line for attenuation correction
			Forwardprojxz<<<256,512>>>(dev_image, lines, realnlines);
			Backprojxz_ac <<<256, 512 >>> (dev_image, dev_back_image, lines, realnlines, 0);//changed 			
																							//Backprojxz<<<256,512>>>(dev_image,dev_back_image,lines,realnlines,0);
		} // if using OSEM, move the iteration to #OSEM
	
		if(DebugInfo>0)
		{
			printf("(IMAGE) rotated image 90 degrees to point to yz plane\n");
		}
		Frotate<<<256,512>>>(dev_back_image, dev_tempback_image);
		Frotate <<<256, 512 >>> (attenuation_matrix, temp_attenuation_matrix);


		if(DebugInfo>0)
		{
			printf("***********************************************************************************\n");
			printf("Doing forward and backward projection for plane yz with lor hitting yz plane (lor-yz)\n");	
			printf("***********************************************************************************\n");
		}
		hipMemcpy(dev_back_image, dev_tempback_image, Nx*Ny*Nz *sizeof(float ),hipMemcpyDeviceToDevice);
		hipMemcpy(attenuation_matrix, temp_attenuation_matrix, Nx * Ny * Nz * sizeof(float), hipMemcpyDeviceToDevice);

		for (i=0; i<totalnumoflinesyz/nlines; i++)
		{
			int realnlines = nlines;
			int noffset = i*nlines;
			

			convertoloryz<<<256,512>>>(dev_lor_data_array,dev_indexxmax,lines,realnlines,noffset);
			attenucorryz << <256, 512 >> > (lines, realnlines, attenuation_matrix);//new line for attenuation correction

			Forwardprojyz<<<256,512>>>(dev_image, lines, realnlines);
			Backprojyz_ac<<<256,512>>>(dev_image,dev_back_image,lines,realnlines,0);
			
		} // if using OSEM, move the iteration to #OSEM

		Frotate<<<256,512>>>(dev_back_image, dev_tempback_image);
		hipMemcpy(dev_back_image, dev_tempback_image, Nx*Ny*Nz *sizeof(float ),hipMemcpyDeviceToDevice);
		hipMemcpy(attenuation_matrix, temp_attenuation_matrix, Nx * Ny * Nz * sizeof(float), hipMemcpyDeviceToDevice);

		// Frotate<<<256,512>>>(dev_back_image, dev_tempback_image);
		// hipMemcpy(dev_back_image, dev_tempback_image, Nx*Ny*Nz *sizeof(float ),hipMemcpyDeviceToDevice);

		// Frotate<<<256,512>>>(dev_back_image, dev_tempback_image);
		// hipMemcpy(dev_back_image, dev_tempback_image, Nx*Ny*Nz *sizeof(float ),hipMemcpyDeviceToDevice);

		if(shouldNormalize>0)
		{
			Fnorm<<<17,720>>>(dev_image,dev_back_image,dev_norm_image);
		}
		else
		{
			hipMemcpy(dev_image, dev_back_image, Nx*Ny*Nz *sizeof(float ),hipMemcpyDeviceToDevice);
		}

		hipMemset(dev_back_image, 0, Nx*Ny*Nz *sizeof(float ));
		hipMemset(lines, 0, CUDAlor_size * nlines * sizeof(float));
		// #OSEM (indicating OSEM)
	}

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,start,stop);

	SaveImageToFile(dev_image, "image.bin", Nx*Ny*Nz);

	printf("************************************************\n");
	printf("   all done!! elapsed time is %f s\n",elapsedTime/1000.0);	
	printf("************************************************\n");

	
	hipFree(dev_lor_data_array);
	hipFree(dev_image); hipFree(dev_back_image); hipFree(dev_tempback_image); hipFree(lines);
	hipFree(dev_indexxmax); hipFree(dev_indexymax);free(sizen);//hipFree(dev_indexzmax);
	return 0;
}
