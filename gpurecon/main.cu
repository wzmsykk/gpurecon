#include "hip/hip_runtime.h"
#include "headerfiles.h"
#include "include/cmdline.h"

#define GRIDSIZEX 128
#define BLOCKSIZEX 256



int main(int argc, char** argv)
{
//  to run:
//	nvcc -arch=sm_20 presort.cu 
//	./a.out will print usage
	hipError_t cudaerr = hipDeviceSetLimit(hipLimitMallocHeapSize, 1048576ULL * 256);
	int ierr;
	cmdline::parser myparser;
	myparser.footer("\n\n a GPU accelerated 3D PET OSEM image reconsruction tool.\n");
	myparser.add<std::string>("lorfile", 'l', "image LOR filename", true, "");
	myparser.add<std::string>("normfile", 'n', "normalization LOR filename", false, "");
	myparser.add<std::string>("ctmhdfile", 'h', "ct header filename(MHD) for ac correction", false, "");
	myparser.add<std::string>("ctbinfile", 'c', "ct binary filename(BIN) for ac correction", false, "");
	myparser.add<std::string>("outputname", 'o', "output image filename", false, "imageZYX.bin");
	myparser.add<int>("bsize", 'b', "batchsize", false,128*128 );
	myparser.add<int>("niter", 'i', "number of iteration", false, 1);
	myparser.add("ac", 'a', "using attenuation correction");
	myparser.parse_check(argc, argv);

	bool use_ac = myparser.exist("ac"); //�Ƿ�ʹ��˥������
	int iterationCount = myparser.get<int>("niter");		//��������
	int batchsize = myparser.get<int>("bsize");			//���δ�С
	
	char* norm_lor_path = const_cast<char*>(myparser.get<std::string>("normfile").c_str());
	char* lor_path = const_cast<char*>(myparser.get<std::string>("lorfile").c_str());
	char* ct_mhd_path = const_cast<char*>(myparser.get<std::string>("ctmhdfile").c_str());
	char* ct_bin_path = const_cast<char*>(myparser.get<std::string>("ctbinfile").c_str());
	char* output_name = const_cast<char*>(myparser.get<std::string>("outputname").c_str());
	//TO DO
	PrintConfig();

	int totalnumoflines,i;
	int shouldNormalize = 0;//Ч������ TODO
	//if (norm_lor_path != "") shouldNormalize = 1;
	
	
	double totalDeviceMemoryUsed=0;
	float * norm_image = (float *)malloc(sizeof(float)*Nx*Ny*Nz);
	float * dev_norm_image;
	hipMalloc ( ( void**)&dev_norm_image, Nx*Ny*Nz * sizeof(float) );
	totalDeviceMemoryUsed += (double)(4*Nx*Ny*Nz * sizeof(float));
	printf("(MEMORY): allocating normalization, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);
	int numoflinesForNorm=0;
	
	if(shouldNormalize >0)//Ч������
	{
		numoflinesForNorm=GetLines(norm_lor_path);
		printf("Calculating normalization image\n");
		CalcNormImage(norm_image, numoflinesForNorm, norm_lor_path);
		FILE * save_norm_imagey;
		save_norm_imagey = fopen ("norm_image.bin" , "w");
		if (save_norm_imagey == NULL) 
		{
			printf("can not write to image file!\n");
			exit(1);
		}
		fwrite(norm_image, sizeof(float), Nx*Ny*Nz, save_norm_imagey);
		hipMemcpy(dev_norm_image, norm_image, Nx*Ny*Nz *sizeof(float ),hipMemcpyHostToDevice);
		free(norm_image);
	}


	// get number of lines from lor files
	totalnumoflines=GetLines(lor_path);
	if( totalnumoflines <= 0)
	{
		printf("Empty lor file.\n");
		exit(1);
	}

	printf("Num of LORs is: %d\n",totalnumoflines);

	FILE * lor_data;
  	lor_data = fopen(lor_path, "r");
   	if (lor_data == NULL) {
		printf("lor data file %s not found\n", lor_path);
		exit(1);
	}
	else 
	{
		printf("lor data file %s is found\n", lor_path);
	}

	// read data from lor file:
	short *lor_data_array= (short *)malloc(sizeof(short) * totalnumoflines * 6);
	for (i=0;i<totalnumoflines;i++)
	{
		fscanf(lor_data,"%hd\t%hd\t%hd\t%hd\t%hd\t%hd\n",
			&lor_data_array[6*i],
			&lor_data_array[6*i+1],
			&lor_data_array[6*i+2],
			&lor_data_array[6*i+3],
			&lor_data_array[6*i+4],
			&lor_data_array[6*i+5]);
	}

	// copy data from local to device
	short *dev_lor_data_array;

	totalDeviceMemoryUsed += (double)(6*totalnumoflines * sizeof(short));
	printf("(MEMORY): allocating LOR data, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);

	hipMalloc ( ( void**)&dev_lor_data_array, 6*totalnumoflines * sizeof(short) );
	hipMemcpy(dev_lor_data_array, lor_data_array, 6*totalnumoflines *sizeof(short ),hipMemcpyHostToDevice);
	free(lor_data_array);
	
	float * dx_array; float * dy_array; float * dz_array;
	hipMalloc ( ( void**)&dx_array,totalnumoflines*sizeof(float));
	hipMalloc ( ( void**)&dy_array,totalnumoflines*sizeof(float));
	hipMalloc ( ( void**)&dz_array,totalnumoflines*sizeof(float));

	totalDeviceMemoryUsed += (double)(3*totalnumoflines * sizeof(float));
	printf("(MEMORY): allocating delta x, y, z data, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);
	printf("sorting delta x, delta y, delta z\n");
	convertolor<<<256,512>>>(dev_lor_data_array,dx_array,dy_array,dz_array,totalnumoflines);

	float *hx_array= (float *)malloc(sizeof(float)*totalnumoflines);
	float *hy_array= (float *)malloc(sizeof(float)*totalnumoflines);
	float *hz_array= (float *)malloc(sizeof(float)*totalnumoflines);	
	hipMemcpy(hx_array, dx_array, sizeof(float)*totalnumoflines,hipMemcpyDeviceToHost);
	hipMemcpy(hy_array, dy_array, sizeof(float)*totalnumoflines,hipMemcpyDeviceToHost);
	hipMemcpy(hz_array, dz_array, sizeof(float)*totalnumoflines,hipMemcpyDeviceToHost);
	hipFree(dx_array);hipFree(dy_array);hipFree(dz_array);

	totalDeviceMemoryUsed -= (double)(3*totalnumoflines * sizeof(float));
	printf("(MEMORY): de-allocating delta x, y, z data, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);

	int *indexxmax = (int *)malloc(sizeof(int)*totalnumoflines);
	int *indexymax = (int *)malloc(sizeof(int)*totalnumoflines);
	int *indexzmax = (int *)malloc(sizeof(int)*totalnumoflines);
	int *sizen = (int *)malloc(sizeof(int)*3);
	
	partlor(hx_array,hy_array,hz_array, totalnumoflines, indexxmax, indexymax, indexzmax, sizen);	
	free(hx_array);   free(hy_array);	free(hz_array);

	int *dev_indexxmax;	int *dev_indexymax;//	int *dev_indexzmax; 	
	hipMalloc ( ( void**)&dev_indexxmax, sizen[0] * sizeof(int) );
	hipMemcpy(dev_indexxmax, indexxmax, sizen[0] * sizeof(int),hipMemcpyHostToDevice);
	hipMalloc ( ( void**)&dev_indexymax, sizen[1] * sizeof(int) );
	hipMemcpy(dev_indexymax, indexymax, sizen[1] * sizeof(int),hipMemcpyHostToDevice);
	free(indexxmax);   free(indexymax);   free(indexzmax);   	

	totalDeviceMemoryUsed += (double)(sizen[0] * sizeof(int));
	totalDeviceMemoryUsed += (double)(sizen[1] * sizeof(int));
	printf("(MEMORY): allocating xz, yz plane max value, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);

	float *image = (float *)malloc(sizeof(float)*Nx*Ny*Nz);
	for (i=0;i<Nx*Ny*Nz;i++){*(image+i) = 1.0;}
	float * dev_image;
	hipMalloc ( ( void**)&dev_image, Nx*Ny*Nz * sizeof(float) );

	totalDeviceMemoryUsed += (double)(sizeof(float)*Nx*Ny*Nz);
	printf("(MEMORY): allocating output image, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);
	hipMemcpy(dev_image, image, Nx*Ny*Nz *sizeof(float ),hipMemcpyHostToDevice);
	free(image);
	
	float * host_back_image = (float *)malloc(sizeof(float)*Nx*Ny*Nz);
	for (i=0;i<Nx*Ny*Nz;i++){*(host_back_image+i) = 0;}
	float * dev_back_image;
	hipMalloc ( ( void**)&dev_back_image, Nx*Ny*Nz * sizeof(float) );

	float * dev_tempback_image;
	hipMalloc ( ( void**)&dev_tempback_image, Nx*Ny*Nz * sizeof(float) );

	totalDeviceMemoryUsed += (double)(2*sizeof(float)*Nx*Ny*Nz);
	printf("(MEMORY): allocating temp image for back projection, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);
	hipMemcpy(dev_tempback_image, host_back_image, Nx*Ny*Nz *sizeof(float ),hipMemcpyHostToDevice);
	hipMemcpy(dev_back_image, host_back_image, Nx*Ny*Nz *sizeof(float ),hipMemcpyHostToDevice);
	free(host_back_image);


	int nlines = 128*128; // can adjust this one to make recon faster (need more memory)
	nlines=batchsize;
	CUDAlor* lines;
	hipMalloc ( ( void**)&lines, sizeof(CUDAlor) * nlines );	// 11 elements for the lines structure

	totalDeviceMemoryUsed += (double)(sizeof(CUDAlor) * nlines );
	printf("(MEMORY): allocating memory to store temp lor data for forward projection, device memory used: %lf MB\n", totalDeviceMemoryUsed/1048576.0);



	/*˥��������ʼ��*/
	float* device_attenuation_matrix; //�Դ�˥������


	short* host_ct_matrix_short = nullptr;
	short* dev_ct_matrix_short = nullptr; //CT���� TO DO

	CTdims* dev_ctdim,*host_ctdim;			//CT������
	int xdim, ydim, zdim, max_len;


	int bsizeac = nlines;//AC���δ�С ��ʱ�����ؽ����δ�С
		//���������������
	LineStatus* dev_linestat; //LOR stat
	float* dev_amin, * dev_amax;
	float* dev_tempvec_x_4f, * dev_tempvec_y_4f, * dev_tempvec_z_4f;
	float* dev_tempmat_alphas;
	float* dev_mat_alphas;
	float* dev_dis;
	int* dev_alphavecsize;
	VoxelID* dev_voxelidvec;
	//�洢����������
	float* dev_linesxz_attvalue_list, * dev_linesyz_attvalue_list;

	//15��device ����
	//6 ��host ����


	//�����ڴ� ת��CT
	if (use_ac) {


		//device ct��Ϣ�ṹ���ʼ��
		hipMalloc((void**)&dev_ctdim, sizeof(CTdims));
		hipMemset(dev_ctdim, 0, sizeof(CTdims));
		//host ct��Ϣ�ṹ���ʼ��
		host_ctdim = (CTdims*)malloc(sizeof(CTdims));
		memset(host_ctdim, 0, sizeof(CTdims));

		//��mhd�ļ��ж�ȡCT��Ϣ
		ierr = genctdim(host_ctdim, ct_mhd_path);
		if (ierr != 0) //TO DO �ж�err
		{
			exit(ierr);
		}
		

		//�õ�ct��voxel����, ��ʼ��host_attenu_matrix˥������
		size_t ctvoxcount = host_ctdim->xdim * host_ctdim->ydim * host_ctdim->zdim;
		hipMalloc((void**)&device_attenuation_matrix, ctvoxcount * sizeof(float));
		hipMemset(device_attenuation_matrix, 0, ctvoxcount * sizeof(float));

		//ͳ��
		totalDeviceMemoryUsed += (double)(2 * sizeof(float) * ctvoxcount);
		printf("(MEMORY): allocating memory to store temp attenuation matrix, device memory used: %lf MB\n", totalDeviceMemoryUsed / 1048576.0);

		printf("(INFO): converting ct matrix values into attenuation values.\n");
		ierr = genacmatrix(device_attenuation_matrix, host_ctdim, ct_bin_path); //��CT����ת��Ϊ˥��ֵ
		if (ierr != 0)
		{
			exit(ierr);
		}


		printf("(INFO): done.\n");
		if (DebugFile > 0) {
			SaveImageToFile(device_attenuation_matrix, "ATT_IMAGE.bin", ctvoxcount);//����˥�������ļ�
		}
		hipMemcpy(host_ctdim, dev_ctdim, sizeof(CTdims), hipMemcpyDeviceToHost);

		xdim = host_ctdim->xdim;
		ydim = host_ctdim->ydim;
		zdim = host_ctdim->zdim;
		max_len = xdim + ydim + zdim + 3 + 2; //(dim+1) �Լ�ÿdim֮�� ��1����
		free(host_ctdim);

		size_t onelinebuffersize = 0; //ͳ��ÿ��LOR������Դ��С

		
		//�����Դ�
		hipMalloc((void**)&dev_linestat, sizeof(LineStatus) * bsizeac);
		hipMemset((void*)dev_linestat, 0, sizeof(LineStatus) * bsizeac);
		onelinebuffersize += sizeof(LineStatus);
		hipMalloc((void**)&dev_tempvec_x_4f, sizeof(float) * bsizeac * 4);
		onelinebuffersize += sizeof(float);
		hipMalloc((void**)&dev_tempvec_y_4f, sizeof(float) * bsizeac * 4);
		onelinebuffersize += sizeof(float);
		hipMalloc((void**)&dev_tempvec_z_4f, sizeof(float) * bsizeac * 4);
		onelinebuffersize += sizeof(float);
		hipMalloc((void**)&dev_amin, sizeof(float) * bsizeac);
		onelinebuffersize += sizeof(float);
		hipMalloc((void**)&dev_amax, sizeof(float) * bsizeac);
		onelinebuffersize += sizeof(float);
		hipMalloc((void**)&dev_tempmat_alphas, sizeof(float) * bsizeac * max_len);
		hipMemset((void*)dev_tempmat_alphas, 0, sizeof(float) * bsizeac * max_len);
		onelinebuffersize += sizeof(float) * max_len;
		hipMalloc((void**)&dev_voxelidvec, sizeof(VoxelID) * bsizeac * max_len);
		onelinebuffersize += sizeof(VoxelID) * max_len;
		hipMalloc((void**)&dev_dis, sizeof(float) * bsizeac * max_len);
		onelinebuffersize += sizeof(float) * max_len;
		hipMalloc((void**)&dev_alphavecsize, sizeof(int) * bsizeac);
		hipMemset((void*)dev_alphavecsize, 0, sizeof(int) * bsizeac);
		onelinebuffersize += sizeof(int);
		hipMalloc((void**)&dev_mat_alphas, sizeof(float) * bsizeac * max_len);
		hipMemset((void*)dev_mat_alphas, 0, sizeof(float) * bsizeac * max_len);
		onelinebuffersize += sizeof(float) * max_len;
		hipDeviceSynchronize();

		totalDeviceMemoryUsed += (double)(onelinebuffersize * bsizeac);
		printf("(MEMORY): allocating memory to for attenuation calculation, device memory used: %lf MB\n", totalDeviceMemoryUsed / 1048576.0);

	}
	

	
	/*˥��������ʼ������*/


	int totalnumoflinesxz = sizen[1];
	int totalnumoflinesyz = sizen[0];

	printf("\nlor memory are prepared now running OSEM (running batches of %d lors) \n\n", nlines);

	






	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	printf("total iteration: #%d\n",iterationCount);


	int maxxzbatch, maxyzbatch;
	maxxzbatch = ceil(totalnumoflinesxz / (float)nlines);//�������θ���
	maxyzbatch = ceil(totalnumoflinesyz / (float)nlines);
	int realnlines;


	//��ʼ˥������
	if (use_ac) {
		int maxxzbatch_acv, maxyzbatch_acv;
		maxxzbatch_acv = ceil(totalnumoflinesxz / (float)bsizeac);//�������θ���
		maxyzbatch_acv = ceil(totalnumoflinesyz / (float)bsizeac);
	
		hipMalloc((void**)&dev_linesxz_attvalue_list, sizeof(float) * totalnumoflinesxz);
		hipMalloc((void**)&dev_linesyz_attvalue_list, sizeof(float) * totalnumoflinesyz);

		//����XZ���ϵ�˥��ֵ
		printf("***********************************************************************************\n");
		printf("Doing attenuation correction with lor hitting xz plane (lor-xz)\n");
		printf("***********************************************************************************\n");
		for (i = 0; i < maxxzbatch_acv; i++) {
			realnlines = bsizeac;
			if ((i + 1) * bsizeac > totalnumoflinesxz) {
				realnlines = totalnumoflinesxz - i * bsizeac;
				printf("(DEBUG) LAST BATCH XZ LOR SIZE=%d\n", realnlines);
			}//��ֹ��������batchsize batchsize<0������ֵ�bug
			int noffset = i * bsizeac;
			convertolorxz << <256, 512 >> > (dev_lor_data_array, dev_indexymax, lines, realnlines, noffset);//��lor_data��lor����index_y_max����lines
		
			//˥��������ʼ
			bsizeac = realnlines;//������ʵ�ʵ�LOR���� realnlines<=bsizeac

			calc_stat << <GRIDSIZEX, BLOCKSIZEX >> > (lines, bsizeac, dev_linestat);
			alphaextrema << <GRIDSIZEX, BLOCKSIZEX >> > (lines, bsizeac, dev_ctdim, dev_linestat, dev_amin, dev_amax, dev_tempvec_x_4f, dev_tempvec_y_4f);
			alphavecs << <GRIDSIZEX, BLOCKSIZEX >> > (lines, bsizeac, dev_ctdim, dev_linestat, dev_amin, dev_amax, dev_tempmat_alphas, dev_mat_alphas, dev_alphavecsize);
			dist_and_ID_in_voxel << <GRIDSIZEX, BLOCKSIZEX >> > (lines, bsizeac, dev_ctdim, dev_linestat, dev_voxelidvec, dev_dis, dev_mat_alphas, dev_alphavecsize);
			attu_inner_product << <GRIDSIZEX, BLOCKSIZEX >> > (lines, bsizeac, dev_ctdim, device_attenuation_matrix, dev_linestat, dev_voxelidvec, dev_dis, dev_alphavecsize);


			extract_attenu_value_to_list_with_offset << <GRIDSIZEX, BLOCKSIZEX >> > (lines, bsizeac, dev_linesxz_attvalue_list, noffset);
			//�������
			hipMemset((void*)dev_linestat, 0, sizeof(LineStatus) * bsizeac);
			hipMemset((void*)dev_tempmat_alphas, 0, sizeof(float) * bsizeac * max_len);
			hipMemset((void*)dev_alphavecsize, 0, sizeof(int) * bsizeac);
			hipMemset((void*)dev_mat_alphas, 0, sizeof(float) * bsizeac * max_len);
			hipDeviceSynchronize();

			//˥����������

		}
		printf("attu corr for XZ lines done.\n");
		//����YZ���ϵ�˥��ֵ
		printf("***********************************************************************************\n");
		printf("Doing attenuation correction with lor hitting yz plane (lor-yz)\n");
		printf("***********************************************************************************\n");
		for (i = 0; i < maxyzbatch_acv; i++) {
			realnlines = bsizeac;
			if ((i + 1) * bsizeac > totalnumoflinesyz) {
				realnlines = totalnumoflinesyz - i * bsizeac;
				printf("(DEBUG) LAST BATCH YZ LOR SIZE=%d\n", realnlines);
			}//��ֹ��������batchsize batchsize<0������ֵ�bug
			int noffset = i * bsizeac;
			convertolorxz << <256, 512 >> > (dev_lor_data_array, dev_indexxmax, lines, realnlines, noffset);//��lor_data��lor����index_x_max����lines

			//˥��������ʼ
			bsizeac = realnlines;//������ʵ�ʵ�LOR���� realnlines<=bsizeac

			calc_stat << <GRIDSIZEX, BLOCKSIZEX >> > (lines, bsizeac, dev_linestat);
			alphaextrema << <GRIDSIZEX, BLOCKSIZEX >> > (lines, bsizeac, dev_ctdim, dev_linestat, dev_amin, dev_amax, dev_tempvec_x_4f, dev_tempvec_y_4f);
			alphavecs << <GRIDSIZEX, BLOCKSIZEX >> > (lines, bsizeac, dev_ctdim, dev_linestat, dev_amin, dev_amax, dev_tempmat_alphas, dev_mat_alphas, dev_alphavecsize);
			dist_and_ID_in_voxel << <GRIDSIZEX, BLOCKSIZEX >> > (lines, bsizeac, dev_ctdim, dev_linestat, dev_voxelidvec, dev_dis, dev_mat_alphas, dev_alphavecsize);
			attu_inner_product << <GRIDSIZEX, BLOCKSIZEX >> > (lines, bsizeac, dev_ctdim, device_attenuation_matrix, dev_linestat, dev_voxelidvec, dev_dis, dev_alphavecsize);


			extract_attenu_value_to_list_with_offset << <GRIDSIZEX, BLOCKSIZEX >> > (lines, bsizeac, dev_linesyz_attvalue_list, noffset);
			//�������
			hipMemset((void*)dev_linestat, 0, sizeof(LineStatus) * bsizeac);
			hipMemset((void*)dev_tempmat_alphas, 0, sizeof(float) * bsizeac * max_len);
			hipMemset((void*)dev_alphavecsize, 0, sizeof(int) * bsizeac);
			hipMemset((void*)dev_mat_alphas, 0, sizeof(float) * bsizeac * max_len);
			hipDeviceSynchronize();

			//˥����������

		}
		printf("attu corr for YZ lines done.\n");
	}


	//������ά�ؽ�
	for (int iter=0;iter<iterationCount;iter++)
	{
		printf("now iteration: #%d\n", iter);
		if (DebugInfo > 0)
		{
			printf("***********************************************************************************\n");
			printf("Doing forward and backward projection for plane xz with lor hitting xz plane (lor-xz)\n");
			printf("***********************************************************************************\n");
		}

		//TO DO 
		maxxzbatch = ceil(totalnumoflinesxz / (float)nlines);
		//maxxzbatch = 1;
		for (i= 0; i< maxxzbatch; i++)
		{
			
			realnlines = nlines;
			//realnlines = 3;
			if ((i+1) * nlines > totalnumoflinesxz) {
				realnlines = totalnumoflinesxz - i * nlines;
				printf("(DEBUG) LAST BATCH XZ LOR SIZE=%d ",realnlines);
			}//��ֹ��������batchsize batchsize<0������ֵ�bug
			else {
				printf("(DEBUG) BATCH:%d XZ LOR SIZE=%d ", i, realnlines);
			}
			int noffset = i*nlines;
			
			

			if (use_ac) {
				
				convertolorxz_ac << <256, 512 >> > (dev_lor_data_array, dev_indexymax, lines, dev_linesxz_attvalue_list, realnlines, noffset);//��lor_data��lor����index_y����lines
				Forwardprojxz << <256, 512 >> > (dev_image, lines, realnlines);
				Backprojxz_ac << <256, 512 >> > (dev_image, dev_back_image, lines, realnlines, 0);//changed 			
			}					//Backprojxz<<<128,128>>>(dev_image,dev_back_image,lines,realnlines,0);
			else {
				convertolorxz << <256, 512 >> > (dev_lor_data_array, dev_indexymax, lines, realnlines, noffset);//��lor_data��lor����index_y����lines
				Forwardprojxz << <256, 512 >> > (dev_image, lines, realnlines);
				Backprojxz << <256, 512 >> > (dev_image, dev_back_image, lines, realnlines, 0);//changed
			}
			hipDeviceSynchronize();
			printf("Done!\n");
			
		} // if using OSEM, move the iteration to #OSEM
		
		if(DebugInfo>0)
		{
			printf("(IMAGE) rotated image 90 degrees to point to yz plane\n");
		}
		Frotate<<<256,512>>>(dev_back_image, dev_tempback_image);
		hipDeviceSynchronize();
		if(DebugInfo>0)
		{
			printf("***********************************************************************************\n");
			printf("Doing forward and backward projection for plane yz with lor hitting yz plane (lor-yz)\n");	
			printf("***********************************************************************************\n");
		}
		hipMemcpy(dev_back_image, dev_tempback_image, Nx*Ny*Nz *sizeof(float ),hipMemcpyDeviceToDevice);

		maxyzbatch = ceil(totalnumoflinesyz / (float)nlines);
		//maxyzbatch = 1;
		for (i = 0; i < maxyzbatch; i++)
		//for (i = 0; i < totalnumoflinesyz / nlines ; i++)
		{
			
			int realnlines = nlines;
			if ((i + 1) * nlines > totalnumoflinesyz) {
				realnlines = totalnumoflinesyz - i * nlines;
				printf("(DEBUG) LAST BATCH YZ LOR SIZE=%d ", realnlines);
			}//��ֹ��������batchsize
			else {
				printf("(DEBUG) BATCH:%d YZ LOR SIZE=%d ", i, realnlines);
			}
			int noffset = i*nlines;
			

			

			if (use_ac) {

				convertoloryz_ac << <256, 512 >> > (dev_lor_data_array, dev_indexxmax, lines, dev_linesyz_attvalue_list, realnlines, noffset);
				Forwardprojyz << <256, 512 >> > (dev_image, lines, realnlines);
				Backprojyz_ac << <256, 512 >> > (dev_image, dev_back_image, lines, realnlines, 0);
			}
			else {
				convertoloryz << <256, 512 >> > (dev_lor_data_array, dev_indexxmax, lines, realnlines, noffset);
				Forwardprojyz << <256, 512 >> > (dev_image, lines, realnlines);
				Backprojyz << <256, 512 >> > (dev_image, dev_back_image, lines, realnlines, 0);
			}
			hipDeviceSynchronize();
			printf("Done!\n");
		} // if using OSEM, move the iteration to #OSEM

		Brotate<<<256,512>>>(dev_back_image, dev_tempback_image);//ת��ȥ
		hipMemcpy(dev_back_image, dev_tempback_image, Nx*Ny*Nz *sizeof(float ),hipMemcpyDeviceToDevice);

		if(shouldNormalize>0)
		{
			Fnorm<<<17,720>>>(dev_image,dev_back_image,dev_norm_image);
		}
		else
		{
			hipMemcpy(dev_image, dev_back_image, Nx*Ny*Nz *sizeof(float ),hipMemcpyDeviceToDevice);
		}

		hipMemset(dev_back_image, 0, Nx*Ny*Nz *sizeof(float ));
		hipMemset(lines, 0, sizeof(CUDAlor) * nlines );
		// #OSEM (indicating OSEM)
	}

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,start,stop);


	//SaveImageToFile(dev_image, "image.bin", Nx * Ny * Nz); //���ٴ�Ϊ��ZYX��ʽ
	Rrotate << <256, 512 >> > (dev_image, dev_tempback_image);//��ΪZYX��ʽ	
	SaveImageToFile(dev_tempback_image, const_cast<char *>(output_name), Nx * Ny * Nz);
	if (Nz > 2) {
		SaveImageToFile_EX(dev_tempback_image, "imageZYX_M2.bin", Nx* Ny* Nz, Nx* Ny, Nx* Ny* (Nz - 2));//ȥ�������͵ײ���Ƭ֮��Ľ��
	}
	
	printf("************************************************\n");
	printf("   all done!! elapsed time is %f s\n",elapsedTime/1000.0);	
	printf("************************************************\n");

	
	hipFree(dev_lor_data_array);	
	hipFree(dev_image); hipFree(dev_back_image); hipFree(dev_tempback_image); hipFree(lines);
	hipFree(dev_indexxmax); hipFree(dev_indexymax);free(sizen);//hipFree(dev_indexzmax);



	//˥���������
	if (use_ac) {
		
		hipFree(device_attenuation_matrix);
		hipFree(dev_ct_matrix_short);
		hipFree(dev_linestat);
		hipFree(dev_tempvec_x_4f);
		hipFree(dev_tempvec_y_4f);
		hipFree(dev_tempvec_z_4f);
		hipFree(dev_amin);
		hipFree(dev_amax);
		hipFree(dev_tempmat_alphas);
		hipFree(dev_voxelidvec);
		hipFree(dev_dis);
		hipFree(dev_alphavecsize);
		hipFree(dev_mat_alphas);
		hipFree(dev_linesxz_attvalue_list);
		hipFree(dev_linesyz_attvalue_list);
	}

	//����


	return 0;
}
